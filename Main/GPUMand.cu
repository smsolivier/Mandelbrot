#include "hip/hip_runtime.h"
#include "../Mandelbrot.h"

using namespace std; 

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__ void mandKernel(double minRe, double maxIm, 
							double dreal, double dim, 
							int maxIter, double escape, 
							int imageWidth, int imageHeight, 
							int *image, int *iter, 
							double *smooth) {
	int x = blockIdx.x * blockDim.x + threadIdx.x; 
	int y = blockIdx.y * blockDim.y + threadIdx.y; 

	if (x < imageWidth && y < imageHeight) {

		double cre, cim, zre, zim, temp; 

		cim = maxIm - y*(dim); 

		int index = y*imageWidth + x; 
		cre = minRe + x*dreal; 

		zim = 0; 
		zre = 0; 

		bool inSet = 1; 

		int k; 
		for (k=0; k<maxIter; k++) {
			temp = pow(zre,2) - pow(zim, 2) + cre; 
			zim = 2*zre*zim + cim; 
			zre = temp; 

			if (pow(zre,2) + pow(zim,2) > 4) {
				inSet = 0; 
				break; 
			}
		}
		if (inSet) image[index] = 1; 
		else if (!inSet) image[index] = 0; 
		else printf("major problem"); 

		iter[index] = k; 

		double mag = pow(pow(zre,2) + pow(zim,2),.5); 
		smooth[index] = k + 1 - log10f(log10f(mag))/log10f(2); 
	} 
}

class GPUMand : public Mandelbrot {
public:
	GPUMand(string name, int resolution, int printBool); 
private:
	void run(); 
	void end();  

	int ncore = 32; 

	int *gimage, *giter; 
	double *gsmooth; 

	hipDeviceProp_t prop; 

	int progBarWidth = 50; 
}; 

GPUMand::GPUMand(string name, int resolution, int printBool) {
	res = resolution; 
	print = printBool; 
	deckname = name; 

	hipGetDeviceProperties(&prop, 0); 

	cout << "Running GPUMandelbrot on " << prop.name << endl;
	cout << ncore << " threads per block" << endl;

	getDeck(); 

	printOpening(); 

	computeTime_start = clock();
	run(); 
	computeTime_stop = clock();

	if (print == 1) printData(); 

	printSet(); 

	end(); 
}

void GPUMand::run() {
	cout << "calculating set" << endl;
	gpuErrchk(hipMalloc(&gimage, sizeof(int)*imageWidth*imageHeight)); 
	gpuErrchk(hipMalloc(&giter, sizeof(int)*imageWidth*imageHeight)); 
	gpuErrchk(hipMalloc(&gsmooth, sizeof(double)*imageWidth*imageHeight)); 

	size_t total; size_t avail; 
	hipMemGetInfo(&avail, &total); 
	cout << "vram used = " << (total - avail)/1e9 << " GB" << endl;

	dim3 threads(ncore, ncore); 

	int bx = (imageWidth + threads.x - 1)/threads.x; 
	int by = (imageHeight + threads.y - 1)/threads.y; 

	dim3 blocks(bx, by); 
 
	mandKernel<<<blocks, threads>>>(minRe, maxIm, dreal, dim, maxIter, 
		escape, imageWidth, imageHeight, gimage, giter, gsmooth); 

	gpuErrchk(hipMemcpy(image, gimage, 
		sizeof(int)*imageWidth*imageHeight, hipMemcpyDeviceToHost)); 
	gpuErrchk(hipMemcpy(iter, giter, 
		sizeof(int)*imageWidth*imageHeight, hipMemcpyDeviceToHost)); 
	gpuErrchk(hipMemcpy(smooth, gsmooth, 
		sizeof(double)*imageWidth*imageHeight, hipMemcpyDeviceToHost));  

	hipFree(gimage); hipFree(giter); hipFree(gsmooth);  

	// populate hist
	for (int i=0; i<imageHeight; i++) {
		for (int j=0; j<imageWidth; j++) {
			int index = i*imageWidth + j; 
			hist[iter[index]]++; 
		}
	}
}

void GPUMand::end() {
	wallTime_stop = clock(); 

	printTime((computeTime_stop - computeTime_start)/CLOCKS_PER_SEC, "Compute Time = "); 

	printTime((wallTime_stop - wallTime_start)/CLOCKS_PER_SEC, "Wall Time = "); 
}

int main(int argc, char *argv[]) {
	GPUMand g(argv[1], atoi(argv[2]), atoi(argv[3]));
	// GPUMand g("tendril", 1000, 1);  
}